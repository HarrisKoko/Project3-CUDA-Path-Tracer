#include "hip/hip_runtime.h"
#include "interactions.h"

#include "utilities.h"

#include <thrust/random.h>

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine& rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng)); // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__host__ __device__ float schlickFresnel(float cosTheta, float etaI, float etaT) {
    float r0 = (etaI - etaT) / (etaI + etaT);
    r0 = r0 * r0;
    float x = 1.0f - cosTheta;
    return r0 + (1.0f - r0) * powf(x, 5.0f);
}

__host__ __device__ void scatterRay(
    PathSegment& pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material& m,
    thrust::default_random_engine& rng)
{
    glm::vec3 I = glm::normalize(pathSegment.ray.direction);
    glm::vec3 newDir;

    // Emissive: terminate path
    if (m.emittance > 0.0f) {
        pathSegment.color *= m.color * m.emittance;
        pathSegment.remainingBounces = 0;
        return;
    }

    // Reflective
    if (m.hasReflective > 0.0f) {
        newDir = glm::reflect(I, normal);
        pathSegment.ray.origin = intersect + 1e-4f * normal;
        pathSegment.ray.direction = glm::normalize(newDir);
        pathSegment.color *= m.color;
        pathSegment.remainingBounces--;
        return;
    }

    if (m.hasRefractive > 0.0f) {
        float etaI = 1.0f, etaT = m.indexOfRefraction;
        glm::vec3 n = normal;

        bool entering = glm::dot(I, normal) < 0.0f;
        if (!entering) {
            n = -normal;
            etaI = m.indexOfRefraction;
            etaT = 1.0f;
        }

        float cosi = glm::clamp(glm::dot(-I, n), -1.0f, 1.0f);
        float eta = etaI / etaT;

        // Fresnel reflectance
        float F = schlickFresnel(fabsf(cosi), etaI, etaT);

        thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
        float toss = u01(rng);

        float k = 1.0f - eta * eta * (1.0f - cosi * cosi);
        if (k < 0.0f || toss < F) {
            newDir = glm::reflect(I, n);
        }
        else {
            newDir = eta * I + (eta * cosi - sqrtf(k)) * n;

            if (!entering) {
                float dist = intersect.t;  // distance traveled in medium
                glm::vec3 absorb = glm::exp(-m.color * dist);
                pathSegment.color *= absorb;
            }
        }

        // Offset origin along the chosen ray direction (safer for refraction)
        float bias = 1e-4f;
        pathSegment.ray.origin = intersect + bias * newDir;
        pathSegment.ray.direction = glm::normalize(newDir);
        pathSegment.remainingBounces--;
        return;
    }




    // Diffuse
    newDir = calculateRandomDirectionInHemisphere(normal, rng);
    pathSegment.ray.origin = intersect + 1e-4f * normal;
    pathSegment.ray.direction = glm::normalize(newDir);
    pathSegment.color *= m.color;
    pathSegment.remainingBounces--;
}




