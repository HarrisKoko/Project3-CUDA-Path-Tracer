#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define samplesPerPixel 16

// Optimization Toggles
#define STREAM_COMPACTION 0
#define MATERIAL_SORTING 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

// Functor for stream compaction
struct isRayAlive {
    __host__ __device__
        bool operator()(const PathSegment& path) const {
        return path.remainingBounces > 0;
    }
};

// Comparator for sorting intersections by materialId
struct materialsCmp {
    __host__ __device__
        bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
        return a.materialId < b.materialId;
    }
};

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    // We only use the first pixelcount slots, but extra capacity is fine.
    hipMalloc(&dev_paths, pixelcount * samplesPerPixel * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int numSamples)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= cam.resolution.x || y >= cam.resolution.y) return;

    int index = x + y * cam.resolution.x;
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f);
    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;

    thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

    glm::vec3 accumulatedDir(0.0f);

    for (int s = 0; s < numSamples; ++s) {
        float jitterX = u01(rng) - 0.5f;
        float jitterY = u01(rng) - 0.5f;

        float px = (float)x + jitterX;
        float py = (float)y + jitterY;

        glm::vec3 rayDir = glm::normalize(
            cam.view
            - cam.right * cam.pixelLength.x * (px - cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * (py - cam.resolution.y * 0.5f)
        );

        accumulatedDir += rayDir;
    }

    segment.ray.direction = glm::normalize(accumulatedDir / float(numSamples));
}

// Intersections only (no shading)
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms
        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // add more primitives as needed

            // track closest hit
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    int bounces)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths || pathSegments[idx].remainingBounces <= 0) return;

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) { // hit something
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, bounces);
        Material material = materials[intersection.materialId];

        glm::vec3 hitPoint = pathSegments[idx].ray.origin +
            intersection.t * pathSegments[idx].ray.direction;

        scatterRay(pathSegments[idx], hitPoint, intersection.surfaceNormal, material, rng);
    }
    else { // miss
        pathSegments[idx].color = glm::vec3(0.0f);
        pathSegments[idx].remainingBounces = 0;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D for primary rays
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    const int blockSize1d = 128;

    // Generate primary rays
    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths, samplesPerPixel);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount; 
    int num_paths = (int)(dev_path_end - dev_paths);    

    bool iterationComplete = false;
    int bounces = 0;
    bool firstBounce = true;

    while (!iterationComplete)
    {
        // Do stream compaction before material sorting to decrease overhead of material sorting
#if STREAM_COMPACTION
        if (!firstBounce) {
            num_paths = thrust::partition(
                thrust::device,
                dev_paths,
                dev_paths + num_paths,
                isRayAlive()) - dev_paths;
        }
#endif

        if (num_paths <= 0) {
            iterationComplete = true;
            break;
        }

        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            (int)hst_scene->geoms.size(),
            dev_intersections
            );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();

        // Sort rays by material 
#if MATERIAL_SORTING
        thrust::sort_by_key(
            thrust::device,
            dev_intersections,
            dev_intersections + num_paths,
            dev_paths,
            materialsCmp());
#endif

        shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            bounces
            );
        checkCUDAError("shade");

        depth++;
        bounces++;
        firstBounce = false;

        if (depth >= traceDepth) {
            iterationComplete = true;
        }

        if (guiData != NULL) {
            guiData->TracedDepth = depth;
        }
    }

#if STREAM_COMPACTION
    num_paths = (int)(dev_path_end - dev_paths); 
#endif

    dim3 numBlocksPixels = (num_paths + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);
    checkCUDAError("finalGather");

    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
