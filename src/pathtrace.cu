#include "hip/hip_runtime.h"
// pathtrace.cu

#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
//#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define samplesPerPixel 16

// toggles
#define SORT_MATERIAL_ID 0
#define STREAM_COMPACTION 0

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

// Kernels

// write image to PBO
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}
// Globals

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

static glm::vec3* dev_vertices = nullptr;
static uint32_t* dev_indices = nullptr;
static glm::vec3* dev_normals = nullptr;
static int        dev_index_count = 0;

static BVHNode* dev_bvh = nullptr;
static glm::uvec3* dev_triTriplets = nullptr;
static int        dev_triCount = 0;

// Functors

struct isRayAlive {
    __host__ __device__
        bool operator()(const PathSegment& path) const {
        return path.remainingBounces > 0;
    }
};

struct materialsCmp {
    __host__ __device__
        bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
        return a.materialId < b.materialId;
    }
};

// Init

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * samplesPerPixel * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    if (!scene->vertices.empty()) {
        hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(glm::vec3));
        hipMemcpy(dev_vertices, scene->vertices.data(),
            scene->vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    }
    if (!scene->indices.empty()) {
        hipMalloc(&dev_indices, scene->indices.size() * sizeof(uint32_t));
        hipMemcpy(dev_indices, scene->indices.data(),
            scene->indices.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        dev_index_count = static_cast<int>(scene->indices.size());
    }
    if (!scene->normals.empty()) {
        hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
        hipMemcpy(dev_normals, scene->normals.data(),
            scene->normals.size() * sizeof(glm::vec3),
            hipMemcpyHostToDevice);
    }

    if (!scene->bvhNodes.empty()) {
        hipMalloc(&dev_bvh, scene->bvhNodes.size() * sizeof(BVHNode));
        hipMemcpy(dev_bvh, scene->bvhNodes.data(),
            scene->bvhNodes.size() * sizeof(BVHNode), hipMemcpyHostToDevice);
    }
    if (!scene->triIndexTriplets.empty()) {
        hipMalloc(&dev_triTriplets, scene->triIndexTriplets.size() * sizeof(glm::uvec3));
        hipMemcpy(dev_triTriplets, scene->triIndexTriplets.data(),
            scene->triIndexTriplets.size() * sizeof(glm::uvec3), hipMemcpyHostToDevice);
        dev_triCount = (int)scene->triIndexTriplets.size();
    }

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    hipFree(dev_vertices);
    hipFree(dev_normals);
    hipFree(dev_indices);

    hipFree(dev_bvh);
    hipFree(dev_triTriplets);

    checkCUDAError("pathtraceFree");
}

// Primary ray generation

__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int numSamples)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= cam.resolution.x || y >= cam.resolution.y) return;

    int index = x + y * cam.resolution.x;
    PathSegment& segment = pathSegments[index];

    segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f);
    segment.pixelIndex = index;
    segment.remainingBounces = traceDepth;

    thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
    thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);

    glm::vec3 accumulatedDir(0.0f);

    for (int s = 0; s < numSamples; ++s) {
        float jitterX = u01(rng) - 0.5f;
        float jitterY = u01(rng) - 0.5f;

        float px = (float)x + jitterX;
        float py = (float)y + jitterY;

        glm::vec3 rayDir = glm::normalize(
            cam.view
            - cam.right * cam.pixelLength.x * (px - cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * (py - cam.resolution.y * 0.5f)
        );

        accumulatedDir += rayDir;
    }

    segment.ray.direction = glm::normalize(accumulatedDir / float(numSamples));
}

// Intersections

__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    const glm::vec3* vertices,
    const uint32_t* indices,
    int index_count,
    const glm::vec3* normals,
    const BVHNode* bvh,
    const glm::uvec3* triTriplets,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (path_index >= num_paths) return;

    const PathSegment path = pathSegments[path_index];
    const Ray rayW = path.ray;

    float t_min_world = FLT_MAX;
    int   hit_geom_index = -1;
    glm::vec3 best_normalW(0.0f);

    for (int i = 0; i < geoms_size; ++i) {
        const Geom& g = geoms[i];

        float t_obj = -1.0f;
        glm::vec3 hitP_obj, n_obj;
        glm::vec3 thisNormalW;
        bool outside = true;

        if (g.type == CUBE) {
            t_obj = boxIntersectionTest(const_cast<Geom&>(g), rayW, hitP_obj, thisNormalW, outside);
            if (t_obj > 0.f && t_obj < t_min_world) {
                t_min_world = t_obj;
                hit_geom_index = i;
                best_normalW = thisNormalW;
            }
        }
        else if (g.type == SPHERE) {
            t_obj = sphereIntersectionTest(const_cast<Geom&>(g), rayW, hitP_obj, thisNormalW, outside);
            if (t_obj > 0.f && t_obj < t_min_world) {
                t_min_world = t_obj;
                hit_geom_index = i;
                best_normalW = thisNormalW;
            }
        }
        else if (g.type == MESH) {
            Ray rObj;
            rObj.origin = glm::vec3(g.inverseTransform * glm::vec4(rayW.origin, 1.0f));
            rObj.direction = glm::normalize(glm::vec3(g.inverseTransform * glm::vec4(rayW.direction, 0.0f)));

            float best_t_obj = FLT_MAX;
            glm::vec3 best_n_obj(0.0f);
            bool hitMesh = false;

            // Traverse BVH 
            int stack[64]; int sp = 0; stack[sp++] = 0; // assume root at 0
            while (sp) {
                int ni = stack[--sp];
                BVHNode n = bvh[ni];

                float t0, t1;
                if (!intersectAABB(rObj, n.box, t0, t1) || t0 > best_t_obj) continue;

                if (n.triCount > 0) {
                    // Leaf
                    for (int kk = 0; kk < n.triCount; ++kk) {
                        const glm::uvec3 tri = triTriplets[n.firstTri + kk];
                        const glm::vec3& v0 = vertices[tri.x];
                        const glm::vec3& v1 = vertices[tri.y];
                        const glm::vec3& v2 = vertices[tri.z];

                        float tTri, u, v;
                        glm::vec3 nFace;
                        if (intersectTriangleBarycentric(rObj, v0, v1, v2, tTri, u, v, nFace)) {
                            if (tTri > 0.0f && tTri < best_t_obj) {
                                best_t_obj = tTri;
                                hitMesh = true;

                                // Interpolate normal
                                glm::vec3 nObj;
                                if (normals) {
                                    float w = 1.0f - u - v;
                                    nObj = glm::normalize(w * normals[tri.x] + u * normals[tri.y] + v * normals[tri.z]);
                                }
                                else {
                                    nObj = glm::normalize(nFace);
                                }
                                best_n_obj = nObj;
                            }
                        }
                    }
                }
                else {
                    // Internal: push children
                    if (n.right >= 0) stack[sp++] = n.right;
                    if (n.left >= 0) stack[sp++] = n.left;
                }
            }

            // If this mesh was hit, convert to world space and compete globally
            if (hitMesh) {
                const glm::vec3 hitP_obj = rObj.origin + best_t_obj * rObj.direction;
                const glm::vec3 hitP_world = glm::vec3(g.transform * glm::vec4(hitP_obj, 1.0f));

                // World-space t along the (normalized) world ray
                const float t_world = glm::dot(hitP_world - rayW.origin, glm::normalize(rayW.direction));

                if (t_world > 0.0f && t_world < t_min_world) {
                    t_min_world = t_world;
                    hit_geom_index = i;
                    best_normalW = glm::normalize(glm::vec3(g.invTranspose * glm::vec4(best_n_obj, 0.0f)));
                }
            }
        }

    }

    if (hit_geom_index < 0) {
        intersections[path_index].t = -1.0f;
    }
    else {
        intersections[path_index].t = t_min_world;
        intersections[path_index].materialId = geoms[hit_geom_index].materialid;
        intersections[path_index].surfaceNormal = best_normalW;
    }
}

// Shading

__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    int bounces)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths || pathSegments[idx].remainingBounces <= 0) return;

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) {
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, bounces);
        Material material = materials[intersection.materialId];

        glm::vec3 hitPoint = pathSegments[idx].ray.origin +
            intersection.t * pathSegments[idx].ray.direction;

        scatterRay(pathSegments[idx], hitPoint, intersection.surfaceNormal, material, rng);
    }
    else {
        pathSegments[idx].color = glm::vec3(0.0f);
        pathSegments[idx].remainingBounces = 0;
    }
}

// Final gather

__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

// Main pathtrace loop

void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    const int blockSize1d = 128;

    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths, samplesPerPixel);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    bool iterationComplete = false;
    int bounces = 0;
    while (!iterationComplete)
    {
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            (int)hst_scene->geoms.size(),
            dev_vertices,
            dev_indices,
            dev_index_count,
            dev_normals,
            dev_bvh,
            dev_triTriplets,
            dev_intersections);

        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

#if SORT_MATERIAL_ID
        thrust::sort_by_key(thrust::device,
            dev_intersections, dev_intersections + num_paths,
            dev_paths,
            materialsCmp());
#endif

        shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            bounces);
        checkCUDAError("shade");

#if STREAM_COMPACTION
        num_paths = thrust::partition(
            thrust::device,
            dev_paths,
            dev_paths + num_paths,
            isRayAlive()) - dev_paths;
#endif

        if (depth >= traceDepth || num_paths == 0) {
            iterationComplete = true;
        }
        bounces++;

        if (guiData != NULL) {
            guiData->TracedDepth = depth;
        }
    }

#if STREAM_COMPACTION
    num_paths = (int)(dev_path_end - dev_paths);
#endif

    dim3 numBlocksPixels = (num_paths + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);
    checkCUDAError("finalGather");

    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
